#include "hip/hip_runtime.h"
/*
 * cudaRenderer.cu
 *
 *  Created on: Aug 2, 2021
 *      Author: Andre
 */

#include "cudaRenderer.h"

/* TODO:
 * - The contribution of a photon equates to a sum on the pixel(s) it affects. Figure
 * out how to synchronize the contributions across all photons.
 * - Random number generation has to match that of the boost library. In the future,
 * support for the sse rng as well. */

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

// Store symbols here so as to avoid long argument list
// in kernel calls
struct Constants {
    float* image;
    float* random;
};

__constant__ Constants constants;

__global__ void renderPhotons() {
    //int i = threadIdx.x;

}

void CudaRenderer::renderImage(image::SmallImage& img, int numPhotons) {

    setup(img);

    renderPhotons<<<1,numPhotons>>>();
    CUDA_CALL(hipDeviceSynchronize());

}

/* Allocates device data, sends parameters to device and sets up RNG. */
void CudaRenderer::setup(image::SmallImage& img, int numPhotons) {
    CUDA_CALL(hipMalloc((void **)&cudaImage,
                         img.getXRes()*img.getYRes()*img.getZRes()*sizeof(float)));

    genDeviceRandomNumbers();

    /* Send in parameters to device */
    Constants params;
    params.image = cudaImage;
    params.random = cudaRandom;

    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constants), &params, sizeof(Constants)));

}

// TODO: Ensure ordering and offset are the same as boost
/* Generates random numbers on the device. */
void CudaRenderer::genDeviceRandomNumbers(CudaSeedType seed = CudaSeedType(5489)) {
    int num = requiredRandomNumbers(numPhotons);
    CUDA_CALL(hipMalloc((void **)&cudaRandom, num * sizeof(float)));

    CURAND_CALL(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MT19937));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(generator, seed));

    /* Generate reals uniformly between 0.0 and 1.0 */
    CURAND_CALL(hiprandGenerateUniform(generator, cudaRandom, num));
}

CudaRenderer::~CudaRenderer() {

    if (image) {
        delete image;
    }

    if (cudaImage) {
        CURAND_CALL(hiprandDestroyGenerator(generator));

        CUDA_CALL(hipFree(cudaImage));
        CUDA_CALL(hipFree(cudaRandom));
    }

    // No need to free constant memory
}

/* Required amount of random numbers to run the renderPhotons kernel on numPhotons */
unsigned int requiredRandomNumbers(unsigned int numPhotons) {
    return numPhotons * 4;
}
