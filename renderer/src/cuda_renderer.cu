#include "hip/hip_runtime.h"
/*
 * cuda_renderer.cu
 *
 *  Created on: Aug 2, 2021
 *      Author: Andre
 */

#include "cuda_renderer.h"
#include "cuda_vector.cuh"
#include "cuda_utils.cuh"
#include "cuda_scene.cuh"
#include <iostream>
#include <stdio.h>
#include <chrono>

namespace cuda {

// Store symbols here so as to avoid long argument list
// in kernel calls
struct Constants {
    Float *image;
    int x_res;
    int y_res;
    int z_res;

    Float *random;
    Scene *scene;
    Medium *medium;
    Float weight;

    int maxDepth;
    Float maxPathlength;
    bool useDirect;
    bool useAngularSampling;
    int numPhotons;
};

__constant__ Constants d_constants;

__device__ Float Sampler::sample(short &uses) const{
    int idx = gridDim.x * blockDim.x * blockDim.y * blockIdx.y + gridDim.x * blockDim.x * threadIdx.y
        + blockDim.x * blockIdx.x + threadIdx.x;
    //ASSERT(uses < RANDOM_NUMBERS_PER_PHOTON);
    ASSERT(idx * RANDOM_NUMBERS_PER_PHOTON + uses < m_size);
    return m_random[idx * RANDOM_NUMBERS_PER_PHOTON + uses++];
}

__device__ inline Float safeSqrt(Float x) {
    return x > FPCONST(0.0) ? sqrtf(x) : FPCONST(0.0);
}

__device__ inline void reflect(const TVector3<Float> &a, const TVector3<Float> &n,
                               TVector3<Float> &b) {
    b = -FPCONST(2.0)*dot(a, n)*n + a;
}

__device__ inline bool refract(const TVector3<Float> &a, const TVector3<Float> &n,
                        Float eta, TVector3<Float> &b) {
    TVector3<Float> q = dot(a,n)*n;
    TVector3<Float> p = (a-q)/eta;

    if (p.length() > FPCONST(1.0)) {
        reflect(a, n, b);
        return false;
    } else {
        q.normalize();
        q *= safeSqrt(FPCONST(1.0) - p.lengthSquared());
        b = p + q;
        return true;
    }
}

__device__ inline Float fresnelDielectric(Float cosThetaI, Float cosThetaT, Float eta) {
	if (fabsf(eta - FPCONST(1.0)) < M_EPSILON * max(FPCONST(1.0), fabsf(eta))) {
		return FPCONST(0.0);
	} else {
		Float Rs = (cosThetaI - eta * cosThetaT) / (cosThetaI + eta * cosThetaT);
		Float Rp = (cosThetaT - eta * cosThetaI) / (cosThetaT + eta * cosThetaI);

		return FPCONST(0.5) * Rs * Rs + Rp * Rp;
	}
}

__device__ inline void SmoothDielectric::sample(const TVector3<Float> &in, const TVector3<Float> &n,
				Sampler &sampler, TVector3<Float> &out, short &samplerUses) const {
	if (fabsf(m_ior1 - m_ior2) < M_EPSILON) {
		// index matched
		out = in;
	} else {
		Float eta;
		if (dot(in, n) < -M_EPSILON) {
			// entering ior2 from ior1
			eta = m_ior2/m_ior1;
		}
		else {
			// entering ior1 from ior2
			eta = m_ior1/m_ior2;
		}

		TVector3<Float> outT;
		if (!refract(in, n, eta, outT)) {
			// TIR
			out = outT;
		} else {
			TVector3<Float> outR;
			reflect(in, n, outR);

			Float cosI = absDot(n, in), cosT = absDot(n, outT);
			Float fresnelR = fresnelDielectric(cosI, cosT, eta);

			// return either refracted or reflected direction based on the Fresnel term
			out = (sampler(samplerUses) < fresnelR ? outR : outT);
		}
	}
}

// Sample random ray
__device__ bool AreaTexturedSource::sampleRay(TVector3<Float> &pos, TVector3<Float> &dir,
                                              Float &totalDistance, Sampler& sampler, short &samplerUses) const{
    pos = *m_origin;

    // sample pixel position first
	int pixel = m_textureSampler->sample(sampler(samplerUses));
	int p[2];
	m_texture->ind2sub(pixel, p[0], p[1]);

	// Now find a random location on the pixel
	for (int iter = 1; iter < m_origin->dim; ++iter) {
		pos[iter] += - (*m_plane)[iter - 1] / FPCONST(2.0) +
            p[iter - 1] * (*m_pixelsize)[iter-1] + sampler(samplerUses) * (*m_pixelsize)[iter - 1];
	}

	dir = *m_dir;

	//FIXME: Hack: Works only for m_dir = [-1 0 0]
	Float z   = sampler(samplerUses)*(1-m_ct) + m_ct;
	Float zt  = sqrtf(FPCONST(1.0)-z*z);
	Float phi = sampler(samplerUses)*2*M_PI;
	dir[0] = -z;
	dir[1] = zt*cosf(phi);
	dir[2] = zt*sinf(phi);
	return propagateTillMedium(pos, dir, totalDistance);
}


__device__ inline Float getMoveStep(const Medium *medium, short &uses) {
    Sampler &sampler = *d_constants.scene->sampler;
    return -medium->getMfp() * logf(sampler(uses));
}

__device__ void Scene::er_step(TVector3<Float> &p, TVector3<Float> &d, Float stepSize, Float scaling) const{
#ifndef OMEGA_TRACKING
    d += HALF * stepSize * dV(p, d, scaling);
    p +=        stepSize * d/m_us->RIF(p, scaling);
    d += HALF * stepSize * dV(p, d, scaling);
#else
    Float two = 2; // To avoid type conversion

    TVector3<Float> K1P = stepSize * dP(d);
    TVector3<Float> K1O = stepSize * dOmega(p, d);

    TVector3<Float> K2P = stepSize * dP(d + HALF*K1O);
    TVector3<Float> K2O = stepSize * dOmega(p + HALF*K1P, d + HALF*K1O);

    TVector3<Float> K3P = stepSize * dP(d + HALF*K2O);
    TVector3<Float> K3O = stepSize * dOmega(p + HALF*K2P, d + HALF*K2O);

    TVector3<Float> K4P = stepSize * dP(d + K3O);
    TVector3<Float> K4O = stepSize * dOmega(p + K3P, d + K3O);

    p = p + ONE_SIXTH * (K1P + two*K2P + two*K3P + K4P);
    d = d + ONE_SIXTH * (K1O + two*K2O + two*K3O + K4O);
#endif
}

__device__ void Scene::traceTillBlock(TVector3<Float> &p, TVector3<Float> &d, Float dist, Float &disx, Float &disy, Float &totalOpticalDistance, Float scaling) const{
	TVector3<Float> oldp, oldd;

    Float distance = 0;
    long int maxsteps = dist/m_us->er_stepsize + 1, i, precision = m_us->getPrecision();

    Float current_stepsize = m_us->er_stepsize;

    for(i = 0; i < maxsteps; i++){
    	oldp = p;
    	oldd = d;

    	er_step(p, d, current_stepsize, scaling);

    	// check if we are at the intersection or crossing the sampled dist, then, estimate the distance and keep going more accurately towards the boundary or sampled dist
    	if(!m_block->inside(p) || (distance + current_stepsize) > dist){
    		precision--;
    		if(precision < 0)
    			break;
    		p = oldp;
    		d = oldd;
    		current_stepsize = current_stepsize / 10;
    		i  = 0;
    		maxsteps = 11;
    	}else{
    		distance += current_stepsize;
#if !USE_SIMPLIFIED_TIMING
    		totalOpticalDistance += current_stepsize * m_us->RIF(p, scaling);
#endif
    	}
    }

    //ASSERT(i < maxsteps);
    disx = 0;
    disy = distance;
}

__device__ TVector3<Float> squareToUniformHemisphere(const TVector2<Float> &sample) {
	Float z = sample.y;
	Float r = sqrtf(FPCONST(1.0) - z*z);
	Float sinPhi, cosPhi;
	sincosf(FPCONST(2.0) * M_PI * sample.x, &sinPhi, &cosPhi);
	return TVector3<Float>(r * cosPhi, r * sinPhi, z);
}

__device__ void sampleRandomDirection(TVector3<Float> &randDirection, Sampler &sampler, short &uses){
	randDirection = squareToUniformHemisphere(TVector2<Float>(sampler(uses), sampler(uses))); // this sampling is done in z=1 direction. need to compensate for it.
	Float temp = randDirection.x;
	randDirection.x =-randDirection.z; // compensating that the direction of photon propagation is -x
	randDirection.z = randDirection.y;
	randDirection.y = temp;
}

__device__ void Scene::addEnergyInParticle(const TVector3<Float> &p, const TVector3<Float> &d, Float distTravelled,
                                           int &depth, Float val, Sampler &sampler, short &uses, const Float &scaling) const {

	TVector3<Float> p1 = p;

	TVector3<Float> dirToSensor;

	if( (p.x-m_camera->getOrigin().x) < 1e-4) // Hack to get rid of inf problems for direct connection
		return;

	sampleRandomDirection(dirToSensor, sampler, uses); // Samples by assuming that the sensor is in +x direction.

//#ifdef PRINT_DEBUGLOG
//	std::cout << "dirToSensor: (" << dirToSensor.x << ", " << dirToSensor.y << ", " << dirToSensor.z << ") \n";
//#endif

#ifndef OMEGA_TRACKING
	dirToSensor *= getMediumIor(p1, scaling);
#endif

	Float distToSensor;
	if(!movePhotonTillSensor(p1, dirToSensor, distToSensor, distTravelled, sampler, uses, scaling))
		return;

//#ifdef OMEGA_TRACKING
	dirToSensor.normalize();
//#endif

	TVector3<Float> refrDirToSensor = dirToSensor;
	Float fresnelWeight = FPCONST(1.0);
	Float ior = getMediumIor(p1, scaling);

	if (ior > FPCONST(1.0)) {
		refrDirToSensor.x = refrDirToSensor.x/ior;
		refrDirToSensor.normalize();
//#ifdef PRINT_DEBUGLOG
//        std::cout << "refrDir: (" << refrDirToSensor[0] << ", " <<  refrDirToSensor[1] << ", " << refrDirToSensor[2] << ");" << std::endl;
//#endif
#ifndef USE_NO_FRESNEL
		fresnelWeight = (FPCONST(1.0) -
		fresnelDielectric(dirToSensor.x, refrDirToSensor.x,
			FPCONST(1.0) / ior))
			/ ior / ior;
#endif
	}
	Float foreshortening = dot(refrDirToSensor, m_camera->getDir())/dot(dirToSensor, m_camera->getDir());
	ASSERT(foreshortening >= FPCONST(0.0));

#if USE_SIMPLIFIED_TIMING
	Float totalOpticalDistance = (distTravelled + distToSensor) * m_ior;
#else
	Float totalOpticalDistance = distTravelled;
#endif

	Float distanceToSensor = 0;
	if(!m_camera->propagateTillSensor(p1, refrDirToSensor, distanceToSensor))
		return;
	totalOpticalDistance += distanceToSensor;

    Medium *medium = d_constants.medium;

	Float totalPhotonValue = val*(2*M_PI)
			* expf(-medium->getSigmaT() * distToSensor)
			* medium->getPhaseFunction()->f(d/d.length(), dirToSensor) // FIXME: Should be refractive index
			* foreshortening
			* fresnelWeight;
	addEnergyToImage(p1, totalOpticalDistance, depth, totalPhotonValue);
//#ifdef PRINT_DEBUGLOG
//    std::cout << "Added Energy:" << totalPhotonValue << " to (" << p1.x << ", " << p1.y << ", " << p1.z << ") at time:" << totalOpticalDistance << std::endl;
//    std::cout << "val term:" << val << std::endl;
//    std::cout << "exp term:" << std::exp(-medium.getSigmaT() * distToSensor) << std::endl;
//    std::cout << "phase function term:" << medium.getPhaseFunction()->f(d/d.length(), dirToSensor) << std::endl;
//    std::cout << "fresnel weight:" << fresnelWeight << std::endl;
//#endif
}

__device__ bool Scene::movePhotonTillSensor(TVector3<Float> &p, TVector3<Float> &d, Float &distToSensor, Float &totalOpticalDistance,
                                            Sampler &sampler, short& uses, const Float& scaling) const {

	Float LargeDist = FPCONST(10000.0);

	Float disx, disy;
	TVector3<Float> d1, norm;
	traceTillBlock(p, d, LargeDist, disx, disy, totalOpticalDistance, scaling);
	distToSensor = disy;
	LargeDist -= disy;
	while(true){
		if(LargeDist < 0){
			//std::cout << "Error in movePhotonTillSensorCode; Large distance is not large enough" << std::endl;
			return false;
		}
		int i;
		norm.zero();
		for (i = 0; i < p.dim; ++i) {
			if (fabsf(m_block->getBlockL()[i] - p[i]) < 2*M_EPSILON) {
				norm[i] = -FPCONST(1.0);
				break;
			}
			else if (fabsf(m_block->getBlockR()[i] - p[i]) < 2*M_EPSILON) {
				norm[i] = FPCONST(1.0);
				break;
			}
		}
		//ASSERT(i < p.dim);

		Float minDiff = M_MAX;
		Float minDir = FPCONST(0.0);
		TVector3<Float> normalt;
		normalt.zero();
		int chosenI = p.dim;
		for (i = 0; i < p.dim; ++i) {
			Float diff = fabsf(m_block->getBlockL()[i] - p[i]);
			if (diff < minDiff) {
				minDiff = diff;
				chosenI = i;
				minDir = -FPCONST(1.0);
			}
			diff = fabsf(m_block->getBlockR()[i] - p[i]);
			if (diff < minDiff) {
				minDiff = diff;
				chosenI = i;
				minDir = FPCONST(1.0);
			}
		}
		normalt[chosenI] = minDir;
		//ASSERT(normalt == norm);
		norm = normalt; // A HACK

        // check if we hit the sensor plane
		if(fabsf(m_camera->getDir().x - norm.x) < M_EPSILON &&
				fabsf(m_camera->getDir().y - norm.y) < M_EPSILON &&
				fabsf(m_camera->getDir().z - norm.z) < M_EPSILON)
			return true;

		// if not, routine
        m_bsdf->sample(d, norm, sampler, d1, uses);
		if (dot(d1, norm) < FPCONST(0.0)) {
			// re-enter the medium through reflection
			d = d1;
		} else {
			return false;
		}

    	traceTillBlock(p, d, LargeDist, disx, disy, totalOpticalDistance, scaling);
    	distToSensor += disy;
    	LargeDist -= disy;
	}

	return true;
}

__device__ inline void addPixel(int x, int y, int z, Float val) {
    Float *image = d_constants.image;
    int x_res = d_constants.x_res;
    int y_res = d_constants.y_res;
    int z_res = d_constants.z_res;

    if (x >= 0 && x < x_res && y >= 0 && y < y_res &&
        z >= 0 && z < z_res) {
        // atomicAdd is atomic within compute device.
        // For coherence with CPU/multiple GPUs, use atomicAdd_system
        atomicAdd(image + (z * x_res * y_res + y * x_res + x), val);
    }
}

__device__ void Scene::addEnergyToImage(const TVector3<Float> &p, Float pathlength, int &depth, Float val) const {

    //printf("Running addEnergyToImage(p = (%.2f, %.2f, %.2f), pathlength = %.2f, depth = %d, val = %.2f)\n", p.x, p.y, p.z, pathlength, depth, val);

	Float x = dot(m_camera->getHorizontal(), p) - m_camera->getOrigin().y;
	Float y = dot(m_camera->getVertical(), p) - m_camera->getOrigin().z;

	//ASSERT(((fabsf(x) < FPCONST(0.5) * m_camera->getPlane().x)
	//			&& (fabsf(y) < FPCONST(0.5) * m_camera->getPlane().y)));
	if (((m_camera->getPathlengthRange().x == -1) && (m_camera->getPathlengthRange().y == -1)) ||
		((pathlength > m_camera->getPathlengthRange().x) && (pathlength < m_camera->getPathlengthRange().y))) {
		x = (x / m_camera->getPlane().x + FPCONST(0.5)) * static_cast<Float>(d_constants.x_res);
		y = (y / m_camera->getPlane().y + FPCONST(0.5)) * static_cast<Float>(d_constants.y_res);

		int ix = static_cast<int>(floorf(x));
		int iy = static_cast<int>(floorf(y));

		int iz;
		if(m_camera->isBounceDecomposition()){
			iz = depth;
		}
		else{
			if ((m_camera->getPathlengthRange().x == -1) && (m_camera->getPathlengthRange().y == -1)) {
				iz = 0;
			} else {
				Float z = pathlength - m_camera->getPathlengthRange().x;
				Float range = m_camera->getPathlengthRange().y - m_camera->getPathlengthRange().x;
				z = (z / range) * static_cast<Float>(d_constants.z_res);
				iz = static_cast<int>(floorf(z));
			}
		}
#ifdef USE_PIXEL_SHARING
		Float fx = x - floorf(x);
		Float fy = y - floorf(y);

		addPixel(ix, iy, iz, val*(FPCONST(1.0) - fx)*(FPCONST(1.0) - fy));
		addPixel(ix + 1, iy, iz, val*fx*(FPCONST(1.0) - fy));
		addPixel(ix, iy + 1, iz, val*(FPCONST(1.0) - fx)*fy);
		addPixel(ix + 1, iy + 1, iz, val*fx*fy);
#else
		addPixel(ix, iy, iz, val);
#endif
    }
}

// Move photon and return true if still in medium, false otherwise
__device__ bool Scene::movePhoton(TVector3<Float> &p, TVector3<Float> &d, Float dist,
                                  Float &totalOpticalDistance, short &uses, Float scaling) const{

	// Algorithm
	// 1. Move till you reach the boundary or till the distance is reached.
	// 2. If you reached the boundary, reflect with probability and keep progressing TODO: change to weight


	Float disx, disy;
	TVector3<Float> d1, norm;
	traceTillBlock(p, d, dist, disx, disy, totalOpticalDistance, scaling);

	dist -= disy;

	while(dist > M_EPSILON){
		int i;
		norm.zero();
		for (i = 0; i < p.dim; ++i) {
			if (fabsf(m_block->getBlockL()[i] - p[i]) < M_EPSILON) {
				norm[i] = -FPCONST(1.0);
				break;
			}
			else if (fabsf(m_block->getBlockR()[i] - p[i]) < M_EPSILON) {
				norm[i] = FPCONST(1.0);
				break;
			}
		}
		//ASSERT(i < p.dim);

		Float minDiff = M_MAX;
		Float minDir = FPCONST(0.0);
		TVector3<Float> normalt;
		normalt.zero();
		int chosenI = p.dim;
		for (i = 0; i < p.dim; ++i) {
			Float diff = fabsf(m_block->getBlockL()[i] - p[i]);
			if (diff < minDiff) {
				minDiff = diff;
				chosenI = i;
				minDir = -FPCONST(1.0);
			}
			diff = fabsf(m_block->getBlockR()[i] - p[i]);
			if (diff < minDiff) {
				minDiff = diff;
				chosenI = i;
				minDir = FPCONST(1.0);
			}
		}
		normalt[chosenI] = minDir;
		//ASSERT(normalt == norm);
		norm = normalt;

		/*
		 * TODO: I think that, because we always return to same medium (we ignore
		 * refraction), there is no need to adjust radiance by eta*eta.
		 */
		Float magnitude = d.length();
#ifdef PRINT_DEBUGLOG
		std::cout << "Before BSDF sample, d: (" << d.x/magnitude << ", " << d.y/magnitude <<  ", " << d.z/magnitude << "); \n "
				"norm: (" << norm.x << ", " << norm.y << ", " << norm.z << ");" << "A Sampler: " << sampler() << "\n";
#endif
        m_bsdf->sample(d/magnitude, norm, *sampler, d1, uses);
        if (dot(d1, norm) < FPCONST(0.0)) {
			// re-enter the medium through reflection
			d = d1*magnitude;
		} else {
			return false;
		}

    	traceTillBlock(p, d, dist, disx, disy, totalOpticalDistance, scaling);
    	dist -= disy;
	}
	return true;
}

__device__ bool scatterOnce(TVector3<Float> &p, TVector3<Float> &d, Float &dist,
                            Float &totalOpticalDistance, Sampler &sampler, short &samplerUses, const Float &scaling) {
    Medium *medium = d_constants.medium;
    Scene *scene = d_constants.scene;

	if ((medium->getAlbedo() > FPCONST(0.0)) && ((medium->getAlbedo() >= FPCONST(1.0)) || (sampler(samplerUses) < medium->getAlbedo()))) {
		TVector3<Float> d1;
		Float magnitude = d.length();
		medium->getPhaseFunction()->sample(d/magnitude, sampler, samplerUses, d1);
		d = magnitude*d1;
		dist = getMoveStep(medium, samplerUses);
		return scene->movePhoton(p, d, dist, totalOpticalDistance, samplerUses, scaling);
	} else {
		dist = FPCONST(0.0);
		return false;
	}
}

__device__ void directTracing(const TVector3<Float> &p, const TVector3<Float> &d, Sampler &sampler, short &uses, const Float &scaling, Float &totalOpticalDistance) {

    const Camera &camera = d_constants.scene->getCamera();

	TVector3<Float> p1 = p;
	TVector3<Float> d1 = d;

	Float distToSensor;
	if(!d_constants.scene->movePhotonTillSensor(p1, d1, distToSensor, totalOpticalDistance, sampler, uses, scaling))
		return;
	Float fresnelWeight = FPCONST(1.0);

#ifndef OMEGA_TRACKING
	d1.normalize();
#endif
	Float ior = d_constants.scene->getMediumIor(p1, scaling);
	TVector3<Float> refrDirToSensor = d1;

	if (ior > FPCONST(1.0)) {
		refrDirToSensor.x = refrDirToSensor.x/ior;
		refrDirToSensor.normalize();
#ifndef USE_NO_FRESNEL
		fresnelWeight = (FPCONST(1.0) -
		fresnelDielectric(d1.x, refrDirToSensor.x,
			FPCONST(1.0) / ior))
			/ ior / ior;
#endif
	}

	Float foreshortening = dot(refrDirToSensor, camera.getDir())/dot(d1, camera.getDir());
	ASSERT(foreshortening >= FPCONST(0.0));

#if USE_SIMPLIFIED_TIMING
	totalDistance = (distToSensor) * ior;
#endif

	Float distanceToSensor = 0;
	if(!camera.propagateTillSensor(p1, refrDirToSensor, distanceToSensor))
		return;
	totalOpticalDistance += distanceToSensor;

	Float totalPhotonValue = d_constants.weight
			* expf(-d_constants.medium->getSigmaT() * distToSensor)
			* fresnelWeight;
	int depth = 0;
	d_constants.scene->addEnergyToImage(p1, totalOpticalDistance, depth, totalPhotonValue);
}

__device__ void scatter(TVector3<Float> &p, TVector3<Float> &d, Float scaling, Float &totalOpticalDistance, short &uses) {
    Scene *scene = d_constants.scene;
    Medium *medium = d_constants.medium;
    Sampler &sampler = *scene->sampler;
	ASSERT(scene->getMediumBlock()->inside(p));

	if ((medium->getAlbedo() > FPCONST(0.0)) && ((medium->getAlbedo() >= FPCONST(1.0)) || (sampler(uses) < medium->getAlbedo()))) {
		TVector3<Float> pos(p), dir(d);

		Float dist = getMoveStep(medium, uses);
		if (!scene->movePhoton(pos, dir, dist, totalOpticalDistance, uses, scaling)) {
			return;
		}

		int depth = 1;
		Float totalDist = dist;
		while ((d_constants.maxDepth < 0 || depth <= d_constants.maxDepth) &&
				(d_constants.maxPathlength < 0 || totalDist <= d_constants.maxPathlength)) {
            ASSERT(d_constants.useAngularSampling);
			if(d_constants.useAngularSampling)
                scene->addEnergyInParticle(pos, dir, totalOpticalDistance, depth, d_constants.weight, sampler, uses, scaling);
//			else
//				scene.addEnergy(img, pos, dir, totalOpticalDistance, depth, weight, medium, sampler, scaling, costFunction, problem, initialization);
			if (!scatterOnce(pos, dir, dist, totalOpticalDistance, sampler, uses, scaling)){
//#ifdef PRINT_DEBUGLOG
//				std::cout << "sampler after failing scatter once:" << sampler() << std::endl;
//#endif
				break;
			}
//#ifdef PRINT_DEBUGLOG
//			std::cout << "sampler after succeeding scatter once:" << sampler() << std::endl;
//
//			std::cout << "dist: " << dist << "\n";
//			std::cout << "pos: (" << pos.x << ", " << pos.y << ", " << pos.z << ", " << "\n";
//			std::cout << "dir: (" << dir.x << ", " << dir.y << ", " << dir.z << ", " << "\n";
//#endif
#if USE_SIMPLIFIED_TIMING
			totalOpticalDistance += dist;
#endif
			++depth;
		}
	}
}

__global__ void renderPhotons() {
    TVector3<Float> pos;
    TVector3<Float> dir;
    Float totalDistance = 0;
    Float scaling = 0;
    short uses = 0;

    Scene *scene = d_constants.scene;
    Sampler &sampler = *scene->sampler;

    int idx = gridDim.x * blockDim.x * blockDim.y * blockIdx.y + gridDim.x * blockDim.x * threadIdx.y + blockDim.x * blockIdx.x + threadIdx.x;

    // FIXME: Checking for numPhotons limit is not necessary as the more photons the merrier.
    if (idx < d_constants.numPhotons) {
        if (scene->genRay(pos, dir, totalDistance, uses)) {
            scaling = max(min(sinf(scene->getUSPhi_min() + scene->getUSPhi_range() * sampler(uses)), scene->getUSMaxScaling()), -scene->getUSMaxScaling());
#ifndef OMEGA_TRACKING
			dir *= scene->getMediumIor(pos, scaling);
#endif
            if (d_constants.useDirect)
                directTracing(pos, dir, sampler, uses, scaling, totalDistance); // Traces and adds direct energy, which is equal to weight * exp( -u_t * path_length);

            scatter(pos, dir, scaling, totalDistance, uses);
        }
    }
}

void CudaRenderer::renderImage(image::SmallImage& target, const med::Medium &medium, const scn::Scene<tvec::TVector3> &scene, int numPhotons) {
    setup(target, medium, scene, numPhotons);

    dim3 threadGrid(16, 16); // Arbitrary choice, total can go up to 1024 on most architectures, 2048 or 4096 on newer ones.
    int threadsPerBlock = threadGrid.x * threadGrid.y;
    int numBlocks = (numPhotons + threadsPerBlock - 1) / (threadsPerBlock);
    int width = 32; // Arbitrary as well

    // N + (W - 1) / W, to ensure we have enough threads as division rounds down
    dim3 blockGrid((numBlocks + width -1) / width, width);

    CUDA_CALL(hipDeviceSynchronize());

    hipEvent_t start, stop;
    float gpu_time = 0.0f;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));

    hipEventRecord(start);

    renderPhotons<<<blockGrid,threadGrid>>>();
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    CUDA_CALL(hipEventElapsedTime(&gpu_time, start, stop));
    std::cout << "Kernel took " << gpu_time << "ms\n";

    CUDA_CALL(hipMemcpy(image, cudaImage,
                         target.getXRes()*target.getYRes()*target.getZRes()*sizeof(Float),
                         hipMemcpyDeviceToHost));

    // Copy back to target. TODO: Just create a class method set pixels that does memcpy
    for (int x=0; x < target.getXRes(); ++x) {
        for (int y=0; y < target.getYRes(); ++y) {
            for (int z=0; z < target.getZRes(); ++z) {
                // Same calculation as addPixel (aka image.addEnergy)
                target.setPixel(x, y, z, image[z * target.getXRes() * target.getYRes() + y * target.getXRes() + x]);
            }
        }
    }

    cleanup();
}

/* Allocates host and device data and sets up RNG. */
//TODO: introduce medium
void CudaRenderer::setup(image::SmallImage& target, const med::Medium &medium, const scn::Scene<tvec::TVector3> &scene, int numPhotons) {
    /* Allocate host memory */
    image = new Float[target.getXRes()*target.getYRes()*target.getZRes()*sizeof(Float)];

    /* Allocate device memory*/
    CUDA_CALL(hipMalloc((void **)&cudaImage,
                         target.getXRes()*target.getYRes()*target.getZRes()*sizeof(Float)));
    CUDA_CALL(hipMemset(cudaImage, 0, target.getXRes()*target.getYRes()*target.getZRes()*sizeof(Float))); // zero out image
    CUDA_CALL(hipMalloc((void **)&cudaRandom, requiredRandomNumbers(numPhotons) * sizeof(Float)));
    cudaScene = Scene::from(scene, cudaRandom, requiredRandomNumbers(numPhotons) * sizeof(Float));
    cudaMedium = Medium::from(medium);

    /* Setup generator. */
    CURAND_CALL(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MT19937));

    /* Send in parameter pointers to device */
    Constants h_constants = {
        .image              = cudaImage,
        .x_res              = target.getXRes(),
        .y_res              = target.getYRes(),
        .z_res              = target.getZRes(),
        .random             = cudaRandom,
        .scene              = cudaScene,
        .medium             = cudaMedium,
        .weight             = getWeight(medium, scene, numPhotons),
        .maxDepth           = maxDepth,
        .maxPathlength      = maxPathlength,
        .useDirect          = useDirect,
        .useAngularSampling = useAngularSampling,
        .numPhotons         = numPhotons
    };

    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_constants), &h_constants, sizeof(Constants)));

    /* Generate random numbers to be used by each thread */
    genDeviceRandomNumbers(requiredRandomNumbers(numPhotons));

    CUDA_CALL(hipDeviceSynchronize());
}

/* Generates random numbers on the device. */
// TODO: currently sequential, compare to result produced by sequential renderer (as opposed to threaded)
void CudaRenderer::genDeviceRandomNumbers(int num, CudaSeedType seed) {
    smp::SamplerSet sampler(1, 0);
    Float *random = new Float[num];
    for (int i = 0; i < num; i++) {
        random[i] = sampler[0]();
    }

    CUDA_CALL(hipMemcpy(cudaRandom, random, sizeof(Float)*num, hipMemcpyHostToDevice));

    delete[] random;

    // TODO: Enable below to make it parallel
    //CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(generator, seed));
    ///* Generate reals uniformly between 0.0 and 1.0 */
    //CURAND_CALL(hiprandGenerateUniform(generator, cudaRandom, num));
}

void CudaRenderer::cleanup() {
    if (image) delete[] image;

    if (generator) CURAND_CALL(hiprandDestroyGenerator(generator));

    // TODO: Free cudaImage, cudaRandom, cudaScene, cudaMedium
}

CudaRenderer::~CudaRenderer() {}

/* Required amount of random numbers to run the renderPhotons kernel on numPhotons */
unsigned int CudaRenderer::requiredRandomNumbers(int numPhotons) {
    return numPhotons * RANDOM_NUMBERS_PER_PHOTON;
}

}
