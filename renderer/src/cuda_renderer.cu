#include "hip/hip_runtime.h"
/*
 * cuda_renderer.cu
 *
 *  Created on: Aug 2, 2021
 *      Author: Andre
 */

#include "cuda_renderer.h"

/* TODO:
 * - The contribution of a photon equates to a sum on the pixel(s) it affects. Figure
 * out how to synchronize the contributions across all photons.
 * - Random number generation has to match that of the boost library. In the future,
 * support for the sse rng as well.
 * - Support for doubles eventually? */

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

// Store symbols here so as to avoid long argument list
// in kernel calls
struct Constants {
    float* image;
    float* random;
};

__constant__ Constants constants;

__global__ void renderPhotons() {
    //int i = threadIdx.x;

}

void CudaRenderer::renderImage() {
    // Generate random numbers to be used by each thread
    genDeviceRandomNumbers(requiredRandomNumbers());

    renderPhotons<<<1,numPhotons>>>();
    CUDA_CALL(hipDeviceSynchronize());

    CUDA_CALL(hipMemcpy(image, cudaImage,
                         img.getXRes()*img.getYRes()*img.getZRes()*sizeof(float),
                         hipMemcpyDeviceToHost));

    // TODO: Write image to target

}

/* Allocates device data, sends parameters to device and sets up RNG. */
void CudaRenderer::setup() {
    /* Allocate device memory*/
    CUDA_CALL(hipMalloc((void **)&cudaImage,
                         img.getXRes()*img.getYRes()*img.getZRes()*sizeof(float)));
    CUDA_CALL(hipMalloc((void **)&cudaRandom, num * sizeof(float)));

    /* Setup generator. */
    CURAND_CALL(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MT19937));

    /* Send in parameters to device */
    Constants params;
    params.image = cudaImage;
    params.random = cudaRandom;

    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constants), &params, sizeof(Constants)));
}

// TODO: Ensure ordering and offset are the same as boost
/* Generates random numbers on the device. */
void CudaRenderer::genDeviceRandomNumbers(int num, CudaSeedType seed = CudaSeedType(5489)) {
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(generator, seed));
    /* Generate reals uniformly between 0.0 and 1.0 */
    CURAND_CALL(hiprandGenerateUniform(generator, cudaRandom, num));
}

CudaRenderer::~CudaRenderer() {

    if (image) {
        delete image;
    }

    if (cudaImage) {
        CURAND_CALL(hiprandDestroyGenerator(generator));

        CUDA_CALL(hipFree(cudaImage));
        CUDA_CALL(hipFree(cudaRandom));
    }

    // No need to free constant memory
}

void CudaRenderer::compareRNGTo(smp::Sampler sampler, int numSamples) {

    float *random = (float *)malloc(numSamples * sizeof(float));
    genDeviceRandomNumbers(numSamples);
    CUDA_CALL(hipMemcpy(random, cudaRandom, numSamples * sizeof(float), hipMemcpyDeviceToHost));

    // TODO: Print 2 columns, one for cuda one for boost
    printf("Cuda\rBoost\n\n");
    for (int i = 0; i < numSamples; i++) {
        printf("%f\n", random[i]);
    }

    return 0;
}

/* Required amount of random numbers to run the renderPhotons kernel on numPhotons */
unsigned int CudaRenderer::requiredRandomNumbers() {
    return numPhotons * 4;
}
