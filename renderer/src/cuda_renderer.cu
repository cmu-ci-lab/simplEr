#include "hip/hip_runtime.h"
/*
 * cuda_renderer.cu
 *
 *  Created on: Aug 2, 2021
 *      Author: Andre
 */

#include "cuda_renderer.h"
#include "cuda_vector.cuh"
#include "cuda_utils.cuh"
#include "cuda_scene.cuh"

namespace cuda {

// Store symbols here so as to avoid long argument list
// in kernel calls
struct Constants {
    Float *image;
    Float *random;
    Scene *scene;
    Medium *medium;
    Float weight;
};

__constant__ Constants d_constants;

__device__ Float Sampler::sample(short &uses) const{
    ASSERT(uses < RANDOM_NUMBERS_PER_PHOTON);
    ASSERT(threadIdx.x * RANDOM_NUMBERS_PER_PHOTON + uses + 1 < m_size);
    return m_random[threadIdx.x * RANDOM_NUMBERS_PER_PHOTON + uses++];
}

// Sample random ray
__device__ bool AreaTexturedSource::sampleRay(TVector3<Float> &pos, TVector3<Float> &dir,
                                      Float &totalDistance, Sampler *sampler, short &samplerUses) const{
    pos = *m_origin;

    // sample pixel position first
	int pixel = m_textureSampler->sample(sampler->sample(samplerUses));
	int p[2];
	m_texture->ind2sub(pixel, p[0], p[1]);

	// Now find a random location on the pixel
	for (int iter = 1; iter < m_origin->dim; ++iter) {
		pos[iter] += - (*m_plane)[iter - 1] / FPCONST(2.0) +
            p[iter - 1] * (*m_pixelsize)[iter-1] + sampler->sample(samplerUses) * (*m_pixelsize)[iter - 1];
	}

	dir = *m_dir;

	//FIXME: Hack: Works only for m_dir = [-1 0 0]
	Float z   = sampler->sample(samplerUses)*(1-m_ct) + m_ct;
	Float zt  = sqrtf(FPCONST(1.0)-z*z);
    // FIXME: FPCONST(M_PI) might be generating complaints here
	Float phi = sampler->sample(samplerUses)*2*M_PI;
    // FIXME: operator[] overload here might be causing issues
	dir[0] = -z;
	dir[1] = zt*cosf(phi);
	dir[2] = zt*sinf(phi);

	return propagateTillMedium(pos, dir, totalDistance);
}

__device__ void scatter() {
//	Assert(scene.getMediumBlock().inside(p));
//
//	if ((medium.getAlbedo() > FPCONST(0.0)) && ((medium.getAlbedo() >= FPCONST(1.0)) || (sampler() < medium.getAlbedo()))) {
//		VectorType<Float> pos(p), dir(d);
//
//		Float dist = getMoveStep(medium, sampler);
//		if (!scene.movePhoton(pos, dir, dist, totalOpticalDistance, sampler, scaling)) {
//			return;
//		}
//
//#ifdef PRINT_DEBUGLOG
//		std::cout << "dist: " << dist << "\n";
//		std::cout << "pos: (" << pos.x << ", " << pos.y << ", " << pos.z << ") " << "\n";
//		std::cout << "dir: (" << dir.x << ", " << dir.y << ", " << dir.z << ") " << "\n";
//#endif
//		int depth = 1;
//		Float totalDist = dist;
//		while ((m_maxDepth < 0 || depth <= m_maxDepth) &&
//				(m_maxPathlength < 0 || totalDist <= m_maxPathlength)) {
//			if(m_useAngularSampling)
//                scene.addEnergyInParticle(img, pos, dir, totalOpticalDistance, depth, weight, medium, sampler, scaling);
//			else
//				scene.addEnergy(img, pos, dir, totalOpticalDistance, depth, weight, medium, sampler, scaling, costFunction, problem, initialization);
//			if (!scatterOnce(pos, dir, dist, scene, medium, totalOpticalDistance, sampler, scaling)){
//#ifdef PRINT_DEBUGLOG
//				std::cout << "sampler after failing scatter once:" << sampler() << std::endl;
//#endif
//				break;
//			}
//#ifdef PRINT_DEBUGLOG
//			std::cout << "sampler after succeeding scatter once:" << sampler() << std::endl;
//
//			std::cout << "dist: " << dist << "\n";
//			std::cout << "pos: (" << pos.x << ", " << pos.y << ", " << pos.z << ", " << "\n";
//			std::cout << "dir: (" << dir.x << ", " << dir.y << ", " << dir.z << ", " << "\n";
//#endif
//#if USE_SIMPLIFIED_TIMING
//			totalOpticalDistance += dist;
//#endif
//			++depth;
//		}
//	}
}

__global__ void renderPhotons() {
    //int i = threadIdx.x;
    //int num_threads = blockDim.x;

    // TODO: Zero out constants.image

    TVector3<Float> pos;
    TVector3<Float> dir;
    Float totalDistance;
    short uses = 0;

    //Float weight = d_constants.weight;
    Scene *scene = d_constants.scene;

    if (scene->genRay(pos, dir, totalDistance, uses)) {
		  float scaling = max(min(sinf(scene->getUSPhi_min() + scene->getUSPhi_range()*scene->sample(uses)), scene->getUSMaxScaling()), -scene->getUSMaxScaling());
    //    Assert(!m_useDirect);
    //    if(m_useDirect)
    //        directTracing(pos, dir, scene, medium, sampler[id], img[id], weight, scaling, totalDistance); // Traces and adds direct energy, which is equal to weight * exp( -u_t * path_length);
    //    scatter(pos, dir, scene, medium, sampler[id], img[id], weight, scaling, totalDistance, *costFunctions[id], problem[id], initializations+id*3);
    }
}

void CudaRenderer::renderImage(image::SmallImage& target, const med::Medium &medium, const scn::Scene<tvec::TVector3> &scene, int numPhotons) {
    setup(target, medium, scene, numPhotons);

    renderPhotons<<<1,numPhotons>>>();
    CUDA_CALL(hipDeviceSynchronize());

    CUDA_CALL(hipMemcpy(image, cudaImage,
                         target.getXRes()*target.getYRes()*target.getZRes()*sizeof(Float),
                         hipMemcpyDeviceToHost));

    target.copyImage(image, target.getXRes()*target.getYRes()*target.getZRes());

    cleanup();
}

/* Allocates host and device data and sets up RNG. */
//TODO: introduce medium
void CudaRenderer::setup(image::SmallImage& target, const med::Medium &medium, const scn::Scene<tvec::TVector3> &scene, int numPhotons) {
    /* Allocate host memory */
    image = new Float[target.getXRes()*target.getYRes()*target.getZRes()*sizeof(Float)];

    /* Allocate device memory*/
    CUDA_CALL(hipMalloc((void **)&cudaImage,
                         target.getXRes()*target.getYRes()*target.getZRes()*sizeof(Float)));
    CUDA_CALL(hipMalloc((void **)&cudaRandom, requiredRandomNumbers(numPhotons) * sizeof(Float)));
    cudaScene = Scene::from(scene, cudaRandom, requiredRandomNumbers(numPhotons) * sizeof(Float));
    cudaMedium = Medium::from(medium);

    /* Setup generator. */
    CURAND_CALL(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MT19937));

    /* Send in parameter pointers to device */
    Constants h_constants;
    h_constants.image = cudaImage;
    h_constants.random = cudaRandom;
    h_constants.scene = cudaScene;
    h_constants.medium = cudaMedium;
    h_constants.weight = getWeight(medium, scene, numPhotons);

    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_constants), &h_constants, sizeof(Constants)));

    /* Generate random numbers to be used by each thread */
    genDeviceRandomNumbers(requiredRandomNumbers(numPhotons));
}

/* Generates random numbers on the device. */
// TODO: currently sequential, compare to result produced by sequential renderer (as opposed to threaded)
void CudaRenderer::genDeviceRandomNumbers(int num, CudaSeedType seed) {
    smp::SamplerSet sampler(1, 0);
    float *random = new float[num];
    for (int i = 0; i < num; i++) {
        random[i] = sampler[0]();
    }

    CUDA_CALL(hipMemcpy(cudaRandom, random, sizeof(float)*num, hipMemcpyHostToDevice));

    delete[] random;

    // TODO: Enable below to make it parallel
    //CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(generator, seed));
    ///* Generate reals uniformly between 0.0 and 1.0 */
    //CURAND_CALL(hiprandGenerateUniform(generator, cudaRandom, num));
}

void CudaRenderer::cleanup() {
    if (image) delete[] image;

    if (generator) CURAND_CALL(hiprandDestroyGenerator(generator));

    // TODO: Free cudaImage, cudaRandom, cudaScene, cudaMedium
}

CudaRenderer::~CudaRenderer() {}

/* Required amount of random numbers to run the renderPhotons kernel on numPhotons */
unsigned int CudaRenderer::requiredRandomNumbers(int numPhotons) {
    return numPhotons * RANDOM_NUMBERS_PER_PHOTON;
}

}
